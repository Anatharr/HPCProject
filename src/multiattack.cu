// multiattack.c
// Starts multiple instances checking if a password can be found in the dictionnary.
// usage : nb_of_processus dictionnary_file shasum_file


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <string.h>

#define DEBUG false
#define WL_BLOCK 1000
#define MAX_LINE_LENGTH 200
#define MAX_SHADOW_LENGTH 5000
#define MAX_HASH_LENGTH 50

__global__ void check_hash(char **wordlist_block_plain, char **wordlist_block_hash, int lines, char **shadow_db, int shadow_count)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < shadow_count)
	{
		char *current_hash = shadow_db[index];

		#if DEBUG
			printf("[Thread - (%d,%d)] Cracking hash %s (%d)\n", blockIdx.x, threadIdx.x, current_hash, index);
		#endif
		for (int i = 0; i < lines; i++)
		{
			#if DEBUG
				printf("[Thread - (%d,%d)] Testing '%s' (%s)\n", blockIdx.x, threadIdx.x, wordlist_block_plain[i], wordlist_block_hash[i]);
			#endif
			bool ok = true;
			for (int v = 0; v < MAX_HASH_LENGTH; v++)
			{
				if (current_hash[v] == '\0' && wordlist_block_hash[i][v] == '\0')
					break;
				if (current_hash[v] != wordlist_block_hash[i][v] || current_hash[v] == '\0' || wordlist_block_hash[i][v] == '\0')
				{
					ok = false;
					break;
				}
			}
			if (ok) {
				printf("[+] FOUND %s for hash %s (shadow_index = %d)\n", wordlist_block_plain[i], current_hash, index);
				break;
			}
		}
	}
}

int main(int argc, char *argv[])
{
	// parsing arguments
	if (argc < 3)
		fprintf(stderr, "Usage: '%s' dictionnary_file shasum_file\n", argv[0]), exit(EXIT_FAILURE);
	char *dict_file = argv[1];
	char *shasum_file = argv[2];

	// opening files
	FILE *shadow_file = fopen(shasum_file, "r");
	FILE *wordlist_file = fopen(dict_file, "r");
	if (shadow_file == NULL || wordlist_file == NULL)
	{
		printf("Error while opening %s file\n", shadow_file == NULL ? "shadow" : "wordlist");
		exit(EXIT_FAILURE);
	}

	/* ------------- Loading shadow db into device ------------- */
	char *shadow_db[MAX_SHADOW_LENGTH];
	char **shadow_dbGPU;
	char buf[MAX_LINE_LENGTH];

	int shadow_count = 0;

	while ((fgets(buf, MAX_LINE_LENGTH, shadow_file)) != NULL)
	{
		buf[strlen(buf) - 1] = '\0'; // remove the trailing newline
#if DEBUG
		printf("address:%p -> %s\n", buf, buf);
#endif
		hipMallocManaged(&shadow_db[shadow_count], strlen(buf));
		hipMemcpy(shadow_db[shadow_count], buf, strlen(buf), hipMemcpyHostToDevice);
		shadow_count++;
	}

	hipMallocManaged(&shadow_dbGPU, shadow_count * sizeof(char *));
	hipMemcpy(shadow_dbGPU, shadow_db, shadow_count * sizeof(char *), hipMemcpyHostToDevice);

	// #if DEBUG
	// 	printf("[DEBUG] Shadow content - head : \n");
	// 	for (int i = 0; i < 10; i++)
	// 	{
	// 		printf("[%i] : %s\n", i, shadow_db[i]);
	// 	}
	// #endif

	/* ------- Optimizing number of threads & blocks based on 0.5 ratio ------ */
	int M = ceil((double)shadow_count / sqrt((shadow_count / 0.5)));
	int T = ceil((double)shadow_count / (double)M);

#if DEBUG
	printf("[DEBUG] Computed values : M=%d ; T=%d\n", M, T);
#endif

	/* ------------- Creating first parrallelisation by dividing wordlist into several blocks (divide & conquer strategy) ------------- */

	int block_counter = 0;
	while (true)
	{
		size_t lines = 0; /** next index to be used with lineBuffer
					(and number of lines already stored)*/
		char *lineBuffer_plain[WL_BLOCK];
		char *lineBuffer_hash[WL_BLOCK];
		char buf[MAX_LINE_LENGTH];
		while (lines < WL_BLOCK && fgets(buf, sizeof(buf), wordlist_file) != NULL)
		{
			char *plain, *hash;
			buf[strlen(buf) - 1] = '\0'; // remove trailing newline
			for (int v = 0; v < MAX_LINE_LENGTH; v++)
			{
				if (buf[v] == '\0')
				{
					printf("ERROR: invalid input line \"%s\" in wordlist\n", buf);
					exit(EXIT_FAILURE);
				}
				if (buf[v] == ' ')
				{
					buf[v] = '\0';
					plain = buf;
					hash = buf + v + 1;
					break;
				}
			}

			hipMallocManaged(&lineBuffer_plain[lines], strlen(plain) * sizeof(char));
			hipMemcpy(lineBuffer_plain[lines], plain, strlen(plain), hipMemcpyHostToDevice);

			hipMallocManaged(&lineBuffer_hash[lines], strlen(hash) * sizeof(char));
			hipMemcpy(lineBuffer_hash[lines], hash, strlen(hash), hipMemcpyHostToDevice);
			lines++;
		}
		if (lines == 0)
			break;

		block_counter++;
#if DEBUG
		printf("[+] Assigned block %d (read %zd lines)\n", block_counter, lines);
#endif
		char **lineBuffer_plainGPU, **lineBuffer_hashGPU;
		hipMallocManaged(&lineBuffer_plainGPU, lines * sizeof(char *));
		hipMemcpy(lineBuffer_plainGPU, lineBuffer_plain, lines * sizeof(char *), hipMemcpyHostToDevice);

		hipMallocManaged(&lineBuffer_hashGPU, lines * sizeof(char *));
		hipMemcpy(lineBuffer_hashGPU, lineBuffer_hash, lines * sizeof(char *), hipMemcpyHostToDevice);

		check_hash<<<M, T>>>(lineBuffer_plainGPU, lineBuffer_hashGPU, lines, shadow_dbGPU, shadow_count);

		// for (int i = 0; i < lines; i++)
		// {
		//     free(lineBuffer[i]);
		// }

		hipDeviceSynchronize();
	}

	return 0;
}
