/*
 * multiattackManaged.cu
 * First version of our program using only cudaMallocManaged(),
 * which is significantly slower than cudaMalloc()
 */


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/types.h>
#include <string.h>
#include <time.h>

#define DEBUG false
#define WL_BLOCK 1000
#define MAX_LINE_LENGTH 200
#define MAX_SHADOW_LENGTH 5000
#define MAX_HASH_LENGTH 50

// Progress bar
#define PBSTR "||||||||||||||||||||||||||||||||||||||||||||||||||||||||||||"

__global__ void check_hash(char **wordlist_block_plain, char **wordlist_block_hash, int lines, char **shadow_db, int shadow_count)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if (index < shadow_count)
	{
		char *current_hash = shadow_db[index];

#if DEBUG
		printf("[Thread - (%d,%d)] Cracking hash %s (%d)\n", blockIdx.x, threadIdx.x, current_hash, index);
#endif
		for (int i = 0; i < lines; i++)
		{
#if DEBUG
			printf("[Thread - (%d,%d)] Testing '%s' (%s)\n", blockIdx.x, threadIdx.x, wordlist_block_plain[i], wordlist_block_hash[i]);
#endif
			bool ok = true;
			for (int v = 0; v < MAX_HASH_LENGTH; v++)
			{
				if (current_hash[v] == '\0' && wordlist_block_hash[i][v] == '\0')
					break;
				if (current_hash[v] != wordlist_block_hash[i][v] || current_hash[v] == '\0' || wordlist_block_hash[i][v] == '\0')
				{
					ok = false;
					break;
				}
			}
			if (ok)
			{
				printf("\u001b[32m[+] FOUND \u001b[33;1m%s\u001b[0;32m for hash %s (shadow_index = %d)\n\u001b[0m", wordlist_block_plain[i], current_hash, index);
				break;
			}
		}
	}
}

int countlines_from_fp(FILE *fp)
{
	// count the number of lines in the file called filename
	int ch = 0;
	int lines = 0;

	if (fp == NULL)
	{
		printf("ERROR OPENING FILE - proceeding");
		fseek(fp, 0, SEEK_SET);
		return 0;
	}

	lines++;
	while ((ch = fgetc(fp)) != EOF)
	{
		if (ch == '\n')
			lines++;
	}
	fseek(fp, 0, SEEK_SET);
	return lines;
}

void updatePBar(int current_block, int total_num_of_block)
{
	double percentage = (double)current_block / total_num_of_block;
	int pb_width = strlen(PBSTR);
	int lpad = (int)(percentage * pb_width);
	int rpad = pb_width - lpad;
	printf("\r\x1b[33;1m Testing wordlist blocks ... [%d/%d] [%.*s%*s]\x1b[0m", current_block, total_num_of_block, lpad, PBSTR, rpad, "");
	fflush(stdout);
}

int main(int argc, char *argv[])
{
	clock_t total_time_beg = clock();
	double parallel_exec_time = 0;
	bool DISABLE_PBAR = false;
	double M_T_RATIO = 0.5;

	// parsing arguments
	bool error_flag = false;
	if (argc < 3)
		error_flag = true;
	else if (argc == 4)
	{
		sscanf(argv[3], "%lf", &M_T_RATIO);
	}
	else if (argc == 5)
	{
		DISABLE_PBAR = (bool)atoi(argv[4]);
	}
	else if (argc > 5)
	{
		error_flag = true;
	}
	if (error_flag)
		fprintf(stderr, "Usage: '%s' dictionnary_file shasum_file\n", argv[0]), exit(EXIT_FAILURE);
	char *dict_file = argv[1];
	char *shasum_file = argv[2];

	/* ------------- Opening wordlist and shadow file ------------- */
	FILE *shadow_file = fopen(shasum_file, "r");
	FILE *wordlist_file = fopen(dict_file, "r");
	int wordlist_n_lines = 0, total_num_of_block = 0;

	if (DISABLE_PBAR == false)
	{
		wordlist_n_lines = countlines_from_fp(wordlist_file);
		total_num_of_block = (wordlist_n_lines / WL_BLOCK) + 1;
	}

	if (shadow_file == NULL || wordlist_file == NULL)
	{
		printf("Error while opening %s file\n", shadow_file == NULL ? "shadow" : "wordlist"),
			exit(EXIT_FAILURE);
	}

	/* ------------- Loading shadow db into device ------------- */
	char *shadow_db[MAX_SHADOW_LENGTH];
	char **shadow_dbGPU;
	char buf[MAX_LINE_LENGTH];
	int shadow_count = 0;

	while ((fgets(buf, MAX_LINE_LENGTH, shadow_file)) != NULL)
	{
		buf[strlen(buf) - 1] = '\0'; // remove the trailing newline
#if DEBUG
		printf("address:%p -> %s\n", buf, buf);
#endif
		hipMallocManaged(&shadow_db[shadow_count], strlen(buf));
		hipMemcpy(shadow_db[shadow_count], buf, strlen(buf), hipMemcpyHostToDevice);
		shadow_count++;
	}

	hipMallocManaged(&shadow_dbGPU, shadow_count * sizeof(char *));
	hipMemcpy(shadow_dbGPU, shadow_db, shadow_count * sizeof(char *), hipMemcpyHostToDevice);

	// #if DEBUG
	// 	printf("[DEBUG] Shadow content - head : \n");
	// 	for (int i = 0; i < 10; i++)
	// 	{
	// 		printf("[%i] : %s\n", i, shadow_db[i]);
	// 	}
	// #endif

	/* ------- Optimizing number of threads & blocks ------ */
	int M = ceil((double)shadow_count / sqrt((shadow_count / M_T_RATIO)));
	int T = ceil((double)shadow_count / (double)M);

#if DEBUG
	printf("[DEBUG] Computed values : M=%d ; T=%d\n", M, T);
#endif

	/* ------------- Creating first parrallelisation by dividing wordlist into several blocks (divide & conquer strategy) ------------- */

	int block_counter = 0;
	while (true)
	{
		if (DISABLE_PBAR == false)
			updatePBar(block_counter, total_num_of_block);

		size_t lines = 0; /** next index to be used with lineBuffer
					(and number of lines already stored)*/
		char *lineBuffer_plain[WL_BLOCK];
		char *lineBuffer_hash[WL_BLOCK];
		char buf[MAX_LINE_LENGTH];
		while (lines < WL_BLOCK && fgets(buf, sizeof(buf), wordlist_file) != NULL)
		{
			char *plain, *hash;
			buf[strlen(buf) - 1] = '\0'; // remove trailing newline
			for (int v = 0; v < MAX_LINE_LENGTH; v++)
			{
				if (buf[v] == '\0')
				{
					printf("ERROR: invalid input line \"%s\" in wordlist\n", buf);
					exit(EXIT_FAILURE);
				}
				if (buf[v] == ' ')
				{
					buf[v] = '\0';
					plain = buf;
					hash = buf + v + 1;
					break;
				}
			}

			hipMallocManaged(&lineBuffer_plain[lines], strlen(plain) * sizeof(char));
			hipMemcpy(lineBuffer_plain[lines], plain, strlen(plain), hipMemcpyHostToDevice);

			hipMallocManaged(&lineBuffer_hash[lines], strlen(hash) * sizeof(char));
			hipMemcpy(lineBuffer_hash[lines], hash, strlen(hash), hipMemcpyHostToDevice);
			lines++;
		}
		if (lines == 0)
			break;

		block_counter++;
#if DEBUG
		printf("[+] Assigned block %d (read %zd lines)\n", block_counter, lines);
#endif
		char **lineBuffer_plainGPU, **lineBuffer_hashGPU;
		hipMallocManaged(&lineBuffer_plainGPU, lines * sizeof(char *));
		hipMemcpy(lineBuffer_plainGPU, lineBuffer_plain, lines * sizeof(char *), hipMemcpyHostToDevice);

		hipMallocManaged(&lineBuffer_hashGPU, lines * sizeof(char *));
		hipMemcpy(lineBuffer_hashGPU, lineBuffer_hash, lines * sizeof(char *), hipMemcpyHostToDevice);

		clock_t parrallel_exec_time_beg = clock();
		check_hash<<<M, T>>>(lineBuffer_plainGPU, lineBuffer_hashGPU, lines, shadow_dbGPU, shadow_count);
		clock_t parrallel_exec_time_end = clock();
		double parallel_instance_time_spent = (double)(parrallel_exec_time_end - parrallel_exec_time_beg) / CLOCKS_PER_SEC;
		parallel_exec_time += parallel_instance_time_spent;  

		hipDeviceSynchronize();
	}

	/* ------------ Benchmarking - writing results to csv ---------- */
	clock_t total_time_end = clock();

	// Computing the times
	double total_exec_time = (double)(total_time_end - total_time_beg) / CLOCKS_PER_SEC;
	//parallel_exec_time
	double serial_exec_time = total_exec_time - parallel_exec_time;

#if DEBUG
	printf("\n[DEBUG] Times : \n- total exec time = %lfs\n- parallel_exec_time = %lfs\n- serial_exec_time = %lfs\n- number of processes = %d\n", total_exec_time, parallel_exec_time, serial_exec_time, M*T);
#endif

	FILE* csv_fp = fopen("./report/benchmark.csv", "a");
	fprintf(csv_fp,"\n%lf, %lf, %lf, %d\n", total_exec_time, parallel_exec_time, serial_exec_time, M*T);
	fclose(csv_fp);
	return 0;
}